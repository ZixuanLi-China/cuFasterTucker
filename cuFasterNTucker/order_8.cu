#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void Update_Parameter_A_SGD_Order_8(const int order,
		const int update_order, const int core_kernel, const int core_dimen,
		int nnz, int **idx_train_len_device, int ***ptr_train_device,
		int ***idx_train_device, type_of_data **value_train_device,
		type_of_data **parameter_a, type_of_data **parameter_b,
		const type_of_data lambda_a,
		type_of_data **intermediate_variables, type_of_data **a_grad_up,
		type_of_data **a_grad_down) {

	int core = core_dimen;
	int worker = block_size / core;
	int lane_id = threadIdx.x % core;
	int local_id = threadIdx.x / core;
	int worker_id = worker * blockIdx.x + local_id;
	int workers = worker * gridDim.x;

	type_of_data intermediate_variables_shared_0[register_size];
	type_of_data intermediate_variables_shared_1[register_size];
	type_of_data intermediate_variables_shared_2[register_size];
	type_of_data intermediate_variables_shared_3[register_size];
	type_of_data intermediate_variables_shared_4[register_size];
	type_of_data intermediate_variables_shared_5[register_size];
	type_of_data intermediate_variables_shared_6[register_size];
	type_of_data parameter_b_shared[register_size];

#pragma unroll
	for (int kernel_index = 0; kernel_index < register_size; kernel_index++) {
		parameter_b_shared[kernel_index] = parameter_b[(update_order + 7)
				% order][kernel_index * core_dimen + lane_id];
	}
	__syncthreads();

	for (int order_index_0 = ptr_train_device[update_order][0][0] + worker_id;
			order_index_0 < ptr_train_device[update_order][0][1];
			order_index_0 += workers) {

		int start_0 = ptr_train_device[update_order][1][order_index_0];
		int end_0 = ptr_train_device[update_order][1][order_index_0 + 1];

		int order_0 = idx_train_device[update_order][0][order_index_0];
		int index_0 = core_kernel * order_0;

#pragma unroll
		for (int kernel_index = 0; kernel_index < register_size;
				kernel_index++) {
			intermediate_variables_shared_0[kernel_index] =
					intermediate_variables[update_order][index_0 + kernel_index];
		}

		for (int order_index_1 = start_0; order_index_1 < end_0;
				order_index_1++) {

			int start_1 = ptr_train_device[update_order][2][order_index_1];
			int end_1 = ptr_train_device[update_order][2][order_index_1 + 1];
			int order_1 = idx_train_device[update_order][1][order_index_1];
			int index_1 = core_kernel * order_1;

#pragma unroll
			for (int kernel_index = 0; kernel_index < register_size;
					kernel_index++) {

				intermediate_variables_shared_1[kernel_index] =
						intermediate_variables[(update_order + 1) % order][index_1
								+ kernel_index];
			}

			for (int order_index_2 = start_1; order_index_2 < end_1;
					order_index_2++) {

				int start_2 = ptr_train_device[update_order][3][order_index_2];
				int end_2 = ptr_train_device[update_order][3][order_index_2 + 1];
				int order_2 = idx_train_device[update_order][2][order_index_2];
				int index_2 = core_kernel * order_2;

#pragma unroll
				for (int kernel_index = 0; kernel_index < register_size;
						kernel_index++) {

					intermediate_variables_shared_2[kernel_index] =
							intermediate_variables[(update_order + 2) % order][index_2
									+ kernel_index];
				}

				for (int order_index_3 = start_2; order_index_3 < end_2;
						order_index_3++) {

					int start_3 =
							ptr_train_device[update_order][4][order_index_3];
					int end_3 = ptr_train_device[update_order][4][order_index_3
							+ 1];
					int order_3 =
							idx_train_device[update_order][3][order_index_3];
					int index_3 = core_kernel * order_3;

#pragma unroll
					for (int kernel_index = 0; kernel_index < register_size;
							kernel_index++) {

						intermediate_variables_shared_3[kernel_index] =
								intermediate_variables[(update_order + 3)
										% order][index_3 + kernel_index];
					}

					for (int order_index_4 = start_3; order_index_4 < end_3;
							order_index_4++) {

						int start_4 =
								ptr_train_device[update_order][5][order_index_4];
						int end_4 =
								ptr_train_device[update_order][5][order_index_4
										+ 1];
						int order_4 =
								idx_train_device[update_order][4][order_index_4];
						int index_4 = core_kernel * order_4;

#pragma unroll
						for (int kernel_index = 0; kernel_index < register_size;
								kernel_index++) {

							intermediate_variables_shared_4[kernel_index] =
									intermediate_variables[(update_order + 4)
											% order][index_4 + kernel_index];
						}

						for (int order_index_5 = start_4; order_index_5 < end_4;
								order_index_5++) {

							int start_5 =
									ptr_train_device[update_order][6][order_index_5];
							int end_5 =
									ptr_train_device[update_order][6][order_index_5
											+ 1];
							int order_5 =
									idx_train_device[update_order][5][order_index_5];
							int index_5 = core_kernel * order_5;

#pragma unroll
							for (int kernel_index = 0;
									kernel_index < register_size;
									kernel_index++) {

								intermediate_variables_shared_5[kernel_index] =
										intermediate_variables[(update_order + 5)
												% order][index_5 + kernel_index];
							}

							for (int order_index_6 = start_5;
									order_index_6 < end_5; order_index_6++) {

								int start_6 =
										ptr_train_device[update_order][7][order_index_6];
								int end_6 =
										ptr_train_device[update_order][7][order_index_6
												+ 1];
								int order_6 =
										idx_train_device[update_order][6][order_index_6];
								int index_6 = core_kernel * order_6;

#pragma unroll
								for (int kernel_index = 0;
										kernel_index < register_size;
										kernel_index++) {

									intermediate_variables_shared_6[kernel_index] =
											intermediate_variables[(update_order
													+ 6) % order][index_6
													+ kernel_index];
								}

								type_of_data gs = 0.0;

#pragma unroll
								for (int kernel_index = 0;
										kernel_index < register_size;
										kernel_index++) {

									type_of_data gs_temp =
											parameter_b_shared[kernel_index];
									gs_temp *=
											intermediate_variables_shared_0[kernel_index];
									gs_temp *=
											intermediate_variables_shared_1[kernel_index];
									gs_temp *=
											intermediate_variables_shared_2[kernel_index];
									gs_temp *=
											intermediate_variables_shared_3[kernel_index];
									gs_temp *=
											intermediate_variables_shared_4[kernel_index];
									gs_temp *=
											intermediate_variables_shared_5[kernel_index];
									gs_temp *=
											intermediate_variables_shared_6[kernel_index];
									gs += gs_temp;
								}

								for (int order_index_7 = start_6;
										order_index_7 < end_6;
										order_index_7++) {

									int order_7 =
											idx_train_device[update_order][7][order_index_7];
									int index_7 = core_dimen * order_7;

									type_of_data p_a_gs =
											parameter_a[(update_order + 7)
													% order][index_7 + lane_id]
													* gs;

									if (core_dimen == 4) {
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												2);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												1);
										p_a_gs = __shfl_sync(mask, p_a_gs, 0,
												4);
									} else if (core_dimen == 8) {
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												4);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												2);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												1);
										p_a_gs = __shfl_sync(mask, p_a_gs, 0,
												8);
									} else if (core_dimen == 16) {
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												8);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												4);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												2);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												1);
										p_a_gs = __shfl_sync(mask, p_a_gs, 0,
												16);
									} else if (core_dimen == 32) {
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												16);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												8);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												4);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												2);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												1);
										p_a_gs = __shfl_sync(mask, p_a_gs, 0);
									}

									atomicAdd(
											&a_grad_up[(update_order + 7)
													% order][index_7 + lane_id],
											value_train_device[update_order][order_index_7]
													* gs);
									atomicAdd(
											&a_grad_down[(update_order + 7)
													% order][index_7 + lane_id],
											p_a_gs * gs
													+ lambda_a
															* parameter_a[(update_order
																	+ 7) % order][index_7
																	+ lane_id]);
								}
							}
						}
					}
				}
			}
		}
	}
}

__global__ void Update_Parameter_B_CSF_Gradient_Order_8(const int order,
		const int update_order, const int core_kernel, const int core_dimen,
		int nnz, int **ptr_train_len_device, int **idx_train_len_device,
		int ***ptr_train_device, int ***idx_train_device,
		type_of_data **value_train_device, type_of_data **parameter_a,
		type_of_data **parameter_b,
		type_of_data **intermediate_variables, type_of_data *b_sum_up,
		type_of_data *b_sum_down) {

	int core = core_dimen;
	int worker = block_size / core;
	int lane_id = threadIdx.x % core;
	int local_id = threadIdx.x / core;
	int worker_id = worker * blockIdx.x + local_id;
	int workers = worker * gridDim.x;

	type_of_data intermediate_variables_shared_0[register_size];
	type_of_data intermediate_variables_shared_1[register_size];
	type_of_data intermediate_variables_shared_2[register_size];
	type_of_data intermediate_variables_shared_3[register_size];
	type_of_data intermediate_variables_shared_4[register_size];
	type_of_data intermediate_variables_shared_5[register_size];
	type_of_data intermediate_variables_shared_6[register_size];
	type_of_data ho_shared[register_size];
	type_of_data parameter_b_shared[register_size];

	type_of_data b_gard_up_temp[register_size];
	type_of_data b_gard_down_temp[register_size];

#pragma unroll
	for (int kernel_index = 0; kernel_index < register_size; kernel_index++) {
		parameter_b_shared[kernel_index] = parameter_b[(update_order + 7)
				% order][kernel_index * core_dimen + lane_id];
	}

#pragma unroll
	for (int core_kernel_index = 0; core_kernel_index < register_size;
			core_kernel_index++) {
		b_gard_up_temp[core_kernel_index] = 0.0;
		b_gard_down_temp[core_kernel_index] = 0.0;
	}
	__syncthreads();

	for (int order_index_0 = ptr_train_device[update_order][0][0] + worker_id;
			order_index_0 < ptr_train_device[update_order][0][1];
			order_index_0 += workers) {

		int start_0 = ptr_train_device[update_order][1][order_index_0];
		int end_0 = ptr_train_device[update_order][1][order_index_0 + 1];

		int order_0 = idx_train_device[update_order][0][order_index_0];
		int index_0 = core_kernel * order_0;

#pragma unroll
		for (int kernel_index = 0; kernel_index < register_size;
				kernel_index++) {

			intermediate_variables_shared_0[kernel_index] =
					intermediate_variables[update_order][index_0 + kernel_index];
		}

		for (int order_index_1 = start_0; order_index_1 < end_0;
				order_index_1++) {

			int start_1 = ptr_train_device[update_order][2][order_index_1];
			int end_1 = ptr_train_device[update_order][2][order_index_1 + 1];
			int order_1 = idx_train_device[update_order][1][order_index_1];
			int index_1 = core_kernel * order_1;

#pragma unroll
			for (int kernel_index = 0; kernel_index < register_size;
					kernel_index++) {

				intermediate_variables_shared_1[kernel_index] =
						intermediate_variables[(update_order + 1) % order][index_1
								+ kernel_index];
			}

			for (int order_index_2 = start_1; order_index_2 < end_1;
					order_index_2++) {

				int start_2 = ptr_train_device[update_order][3][order_index_2];
				int end_2 = ptr_train_device[update_order][3][order_index_2 + 1];
				int order_2 = idx_train_device[update_order][2][order_index_2];
				int index_2 = core_kernel * order_2;

#pragma unroll
				for (int kernel_index = 0; kernel_index < register_size;
						kernel_index++) {

					intermediate_variables_shared_2[kernel_index] =
							intermediate_variables[(update_order + 2) % order][index_2
									+ kernel_index];
				}

				for (int order_index_3 = start_2; order_index_3 < end_2;
						order_index_3++) {

					int start_3 =
							ptr_train_device[update_order][4][order_index_3];
					int end_3 = ptr_train_device[update_order][4][order_index_3
							+ 1];
					int order_3 =
							idx_train_device[update_order][3][order_index_3];
					int index_3 = core_kernel * order_3;

#pragma unroll
					for (int kernel_index = 0; kernel_index < register_size;
							kernel_index++) {

						intermediate_variables_shared_3[kernel_index] =
								intermediate_variables[(update_order + 3)
										% order][index_3 + kernel_index];
					}

					for (int order_index_4 = start_3; order_index_4 < end_3;
							order_index_4++) {

						int start_4 =
								ptr_train_device[update_order][5][order_index_4];
						int end_4 =
								ptr_train_device[update_order][5][order_index_4
										+ 1];
						int order_4 =
								idx_train_device[update_order][4][order_index_4];
						int index_4 = core_kernel * order_4;

#pragma unroll
						for (int kernel_index = 0; kernel_index < register_size;
								kernel_index++) {

							intermediate_variables_shared_4[kernel_index] =
									intermediate_variables[(update_order + 4)
											% order][index_4 + kernel_index];
						}

						for (int order_index_5 = start_4; order_index_5 < end_4;
								order_index_5++) {

							int start_5 =
									ptr_train_device[update_order][6][order_index_5];
							int end_5 =
									ptr_train_device[update_order][6][order_index_5
											+ 1];
							int order_5 =
									idx_train_device[update_order][5][order_index_5];
							int index_5 = core_kernel * order_5;

#pragma unroll
							for (int kernel_index = 0;
									kernel_index < register_size;
									kernel_index++) {

								intermediate_variables_shared_5[kernel_index] =
										intermediate_variables[(update_order + 5)
												% order][index_5 + kernel_index];
							}

							for (int order_index_6 = start_5;
									order_index_6 < end_5; order_index_6++) {

								int start_6 =
										ptr_train_device[update_order][7][order_index_6];
								int end_6 =
										ptr_train_device[update_order][7][order_index_6
												+ 1];
								int order_6 =
										idx_train_device[update_order][6][order_index_6];
								int index_6 = core_kernel * order_6;

#pragma unroll
								for (int kernel_index = 0;
										kernel_index < register_size;
										kernel_index++) {

									intermediate_variables_shared_6[kernel_index] =
											intermediate_variables[(update_order
													+ 6) % order][index_6
													+ kernel_index];
								}

								type_of_data gs = 0.0;

#pragma unroll
								for (int kernel_index = 0;
										kernel_index < register_size;
										kernel_index++) {

									ho_shared[kernel_index] = 1.0f;
									ho_shared[kernel_index] *=
											intermediate_variables_shared_0[kernel_index];
									ho_shared[kernel_index] *=
											intermediate_variables_shared_1[kernel_index];
									ho_shared[kernel_index] *=
											intermediate_variables_shared_2[kernel_index];
									ho_shared[kernel_index] *=
											intermediate_variables_shared_3[kernel_index];
									ho_shared[kernel_index] *=
											intermediate_variables_shared_4[kernel_index];
									ho_shared[kernel_index] *=
											intermediate_variables_shared_5[kernel_index];
									ho_shared[kernel_index] *=
											intermediate_variables_shared_6[kernel_index];
									type_of_data gs_temp =
											parameter_b_shared[kernel_index];
									gs_temp *= ho_shared[kernel_index];
									gs += gs_temp;

								}

								for (int order_index_7 = start_6;
										order_index_7 < end_6;
										order_index_7++) {

									int order_7 =
											idx_train_device[update_order][7][order_index_7];
									int index_7 = core_dimen * order_7;

									type_of_data parameter_a_temp =
											parameter_a[(update_order + 7)
													% order][index_7 + lane_id];

									type_of_data p_a_gs = parameter_a_temp * gs;

									if (core_dimen == 4) {
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												2);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												1);
										p_a_gs = __shfl_sync(mask, p_a_gs, 0,
												4);
									} else if (core_dimen == 8) {
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												4);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												2);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												1);
										p_a_gs = __shfl_sync(mask, p_a_gs, 0,
												8);
									} else if (core_dimen == 16) {
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												8);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												4);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												2);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												1);
										p_a_gs = __shfl_sync(mask, p_a_gs, 0,
												16);
									} else if (core_dimen == 32) {
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												16);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												8);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												4);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												2);
										p_a_gs += __shfl_down_sync(mask, p_a_gs,
												1);
										p_a_gs = __shfl_sync(mask, p_a_gs, 0);
									}

#pragma unroll
									for (int core_kernel_index = 0;
											core_kernel_index < register_size;
											core_kernel_index++) {
										type_of_data aqs = parameter_a_temp
												* ho_shared[core_kernel_index];
										b_gard_up_temp[core_kernel_index] +=
												value_train_device[update_order][order_index_7]
														* aqs;
										b_gard_down_temp[core_kernel_index] +=
												p_a_gs * aqs;
									}
								}
							}
						}
					}
				}
			}
		}
	}
#pragma unroll
	for (int core_kernel_index = 0; core_kernel_index < register_size;
			core_kernel_index++) {

		atomicAdd(
				&b_sum_up[(worker_id % sum_size) * core_kernel * core_dimen
						+ core_kernel_index * core_dimen + lane_id],
				b_gard_up_temp[core_kernel_index]);
		atomicAdd(
				&b_sum_down[(worker_id % sum_size) * core_kernel * core_dimen
						+ core_kernel_index * core_dimen + lane_id],
				b_gard_down_temp[core_kernel_index]);

	}
}

void Update_Parameter_A_8(const int order, int *dimen, const int core_kernel,
		const int core_dimen, int nnz_train, int **idx_train_len_device,
		int ***ptr_train_device, int ***idx_train_device,
		type_of_data **value_train_device,
		type_of_data **parameter_a_device,
		type_of_data **parameter_b_device, const type_of_data lambda_a,
		type_of_data **intermediate_variables,
		type_of_data **a_grad_up,
		type_of_data **a_grad_down, type_of_data **a_grad_up_host_to_device,
		type_of_data **a_grad_down_host_to_device) {

	for (int i = 0; i < order; i++) {

		hipMemset(a_grad_up_host_to_device[i], 0,
				dimen[i] * core_dimen * sizeof(type_of_data));
		hipMemset(a_grad_down_host_to_device[i], 0,
				dimen[i] * core_dimen * sizeof(type_of_data));
	}

	for (int update_order = 0; update_order < order; update_order++) {

		Update_Parameter_A_SGD_Order_8 <<<grid_size, block_size
		>>>(order, 0, core_kernel, core_dimen, nnz_train,
				idx_train_len_device, ptr_train_device, idx_train_device,
				value_train_device, parameter_a_device, parameter_b_device,
				lambda_a, intermediate_variables, a_grad_up, a_grad_down);
		hipDeviceSynchronize();

		int fact_order = (0 + order - 1) % order;

		Update_Parameter_A <<<dimen[fact_order] / (block_size / core_dimen) + 1,
		block_size>>>(fact_order, dimen[fact_order], core_dimen,
				parameter_a_device, a_grad_up, a_grad_down);
		hipDeviceSynchronize();

		Calculate_Intermediate_Variables
				<<<dimen[fact_order] / block_size * core_dimen, block_size>>>(order,
				core_kernel, core_dimen, parameter_a_device, parameter_b_device,
				fact_order, dimen[fact_order], intermediate_variables);
		hipDeviceSynchronize();
	}
}

void Update_Parameter_B_Batch_8(const int order, int *dimen,
		const int core_kernel, const int core_dimen, int nnz,
		int **ptr_train_len_device, int **idx_train_len_device,
		int ***ptr_train_device, int ***idx_train_device,
		type_of_data **value_train_device,
		type_of_data **parameter_a, type_of_data **parameter_b,
		type_of_data lambda_b,
		type_of_data **intermediate_variables) {

	type_of_data *b_sum_up;
	type_of_data *b_grad_up;

	hipMalloc((void**) &b_sum_up,
	sum_size * core_kernel * core_dimen * sizeof(type_of_data));
	hipMalloc((void**) &b_grad_up,
			core_kernel * core_dimen * sizeof(type_of_data));

	type_of_data *b_sum_down;
	type_of_data *b_grad_down;

	hipMalloc((void**) &b_sum_down,
	sum_size * core_kernel * core_dimen * sizeof(type_of_data));
	hipMalloc((void**) &b_grad_down,
			core_kernel * core_dimen * sizeof(type_of_data));

	for (int update_order = 0; update_order < order; update_order++) {

		hipMemset(b_sum_up, 0,
		sum_size * core_kernel * core_dimen * sizeof(type_of_data));
		hipMemset(b_grad_up, 0,
				core_kernel * core_dimen * sizeof(type_of_data));

		hipMemset(b_sum_down, 0,
		sum_size * core_kernel * core_dimen * sizeof(type_of_data));
		hipMemset(b_grad_down, 0,
				core_kernel * core_dimen * sizeof(type_of_data));

		Update_Parameter_B_CSF_Gradient_Order_8
				<<<grid_size,block_size>>>(order, 0, core_kernel, core_dimen,
				nnz, ptr_train_len_device, idx_train_len_device,
				ptr_train_device, idx_train_device, value_train_device,
				parameter_a, parameter_b, intermediate_variables, b_sum_up,
				b_sum_down);
		hipDeviceSynchronize();

		Parameter_B_Gradient_Sum_Up<<<
		core_kernel / (block_size / core_dimen) + 1, block_size>>>(
				core_kernel, core_dimen, b_sum_up, b_grad_up);
		hipDeviceSynchronize();

		Parameter_B_Gradient_Sum_Down<<<
		core_kernel / (block_size / core_dimen) + 1, block_size>>>(
				core_kernel, core_dimen, nnz, b_sum_down, b_grad_down,
				parameter_b, lambda_b, 0);
		hipDeviceSynchronize();

		Update_Parameter_B<<< core_kernel / (block_size / core_dimen) + 1,
		block_size>>>(0, core_kernel, core_dimen, parameter_b, b_grad_up,
				b_grad_down);

		hipDeviceSynchronize();
		int fact_order = (0 + order - 1) % order;
		Calculate_Intermediate_Variables
				<<<dimen[fact_order] / block_size * core_dimen, block_size>>>(order,
				core_kernel, core_dimen, parameter_a, parameter_b, fact_order,
				dimen[fact_order], intermediate_variables);
		hipDeviceSynchronize();
	}

	hipFree(b_sum_up);
	hipFree(b_grad_up);

	hipFree(b_sum_down);
	hipFree(b_grad_down);
}
